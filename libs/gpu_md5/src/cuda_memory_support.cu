#include "cuda_memory_support.cuh"


template<> void GpuMalloc<void>(void**devPtr, size_t size){
    hipError_t error;
    if(( error = hipMalloc(devPtr,  size)) != hipSuccess){
        throw CudaMemoryError(error);
    }
}

template<> void GpuCopy<void>(void*dst, const void* src, size_t size, hipMemcpyKind kind){
     hipError_t error;
    if((error = hipMemcpy(dst, src, size, kind)) != hipSuccess){
        throw  CudaMemoryError(error);
    }
}

gpumemblock::gpumemblock(size_t size) : _size(size) {
  GpuMalloc(&_data, size);
  GpuMalloc(&_gpusize, 1);
  GpuCopy(_gpusize, &_size, 1, hipMemcpyHostToDevice);
}

__device__ uint8_t *gpumemblock::getblock() { return static_cast<uint8_t *>(_data); }

__device__ size_t gpumemblock::get_size() { return *_gpusize; }

__host__ void gpumemblock::copyfrom(void *data) {
  GpuCopy(_data, data, _size, hipMemcpyHostToDevice);
}
__host__ void gpumemblock::copyto(void *data) {
  GpuCopy(data, _data, _size, hipMemcpyDeviceToHost);
}

__device__ void gpumemblock::copyto(gpumemblock &blk) {
  GpuCopy(blk._data, _data, _size, hipMemcpyDeviceToDevice);
}
