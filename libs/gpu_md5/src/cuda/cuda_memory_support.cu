#include "cuda_memory_support.cuh"


template<> void GpuMalloc<void>(void**devPtr, size_t size){
    hipError_t error;
    if(( error = hipMalloc(devPtr,  size)) != hipSuccess){
        throw CudaMemoryError(error);
    }
}

template<> void GpuCopy<void>(void*dst, const void* src, size_t size, hipMemcpyKind kind){
     hipError_t error;
    if((error = hipMemcpy(dst, src, size, kind)) != hipSuccess){
        throw  CudaMemoryError(error);
    }
}
