#include "hip/hip_runtime.h"
#include "md5gpu.cuh"
#include "gpu_string_generator.cuh"
#include "../cuda_manager.hpp"
#include "md5bruter.cuh"

inline __device__ void hexdigest(const uint8_t digest[16], char hex_output[33]) {
    static const char hex_chars[] = "0123456789abcdef";

    for (int i = 0; i < 16; i++) {
        hex_output[i * 2] = hex_chars[(digest[i] >> 4) & 0xF];
        hex_output[i * 2 + 1] = hex_chars[digest[i] & 0xF];
    }
    hex_output[32] = '\0'; // Null-terminate la stringa
}

#define dbgline() printf("process %d line reached %d\n",i,__LINE__);
__device__ bool cmpstr(const char* a, const char* b, size_t size){
    for(size_t i = 0 ; i < size/8; i++){
        if(((uint64_t*)a)[i] != ((uint64_t*)b)[i]){
            return false;
        }
    }
    return true;
}

#define print_request(request)  printf("request: %s %s %d %ld %ld\n",request->target_md5,request->target_found,request->base_str_len,request->address_start,request->address_end);


__global__ void md5_brute_apply(struct md5_bruter_request * request){
    int i = threadIdx.x+blockDim.x*blockIdx.x;
    size_t span = request->address_end - request->address_start;
    if(i < span){
        char sequence[24];
        memset(sequence,0,24);
        GpuStringGenerator gen = new_generator(request->base_str_len);
        assign_address(&gen,request->address_start + i);
        next_sequence(&gen,sequence);
        MD5Gpu algo{sequence,(size_t)gen.currentSequenceLength};
        const uint8_t* digest = algo.getdigest();
        char result[33]{};
        hexdigest(digest,result);
        if(cmpstr(result, request->target_md5, 32)){
            memcpy(request->target_found,sequence,gen.currentSequenceLength);
            request->target_found[gen.currentSequenceLength] = 0;
        }
        free(sequence);
    }
}


__host__ void CheckGpuCondition();

struct md5_bruter_request * dev_request = nullptr;

__host__ hipError_t alloc_request(){
    return hipMalloc(&dev_request, sizeof(md5_bruter_request));
}

__host__ hipError_t free_request(){
    return hipFree(dev_request);
}

__host__ hipError_t copy_request_to_device(struct md5_bruter_request* request){
    return hipMemcpy(dev_request, request, sizeof(md5_bruter_request), hipMemcpyHostToDevice);
}

__host__ hipError_t copy_request_to_host(struct md5_bruter_request * request){
    return hipMemcpy(request, dev_request, sizeof(md5_bruter_request), hipMemcpyDeviceToHost);
}

#define handle(op) error = op; if(error) goto ERROR;


void md5_gpu_brute(struct md5_bruter_request* request, int threads){
    hipError_t error = hipSuccess;
    CudaManager::instance()->select_gpu();
    int span = request->address_end - request->address_start;
    int blocks = ceil(static_cast<double>(span) / threads);
    handle(alloc_request());
    handle(copy_request_to_device(request));
    hipDeviceSynchronize();

    md5_brute_apply<<<threads, blocks>>>(dev_request);
    hipDeviceSynchronize();
   // handle(hipGetLastError());
    handle(copy_request_to_host(request));
    handle(free_request());
    return;
    ERROR:
        printf("error on computing %s \n",hipGetErrorString(error));
        CudaManager::instance()->disable_current_gpu();
        md5_gpu_brute(request, threads);
        return;
}


